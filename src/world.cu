
#include "world.h"

// Reads the details of a world from the specified file
world_t* World_read (FILE* file)
{
	int item_i;
	// Allocate space for the result
	world_t* result = (world_t*) malloc(sizeof(world_t));

	// Read background color of the world
	fscanf(file, "BG %hu %hu %hu\n", &(result->bg.r), &(result->bg.g), &(result->bg.b));
	// Read global ambient brightness
	fscanf(file, "AMB %f\n", &(result->global_ambient));

	// Read the lights
	fscanf(file, "LIGHTS %u\n", &(result->n_lights));
	result->lights = (light_t*) malloc(sizeof(light_t) * result->n_lights);
	for (item_i = 0; item_i < result->n_lights; item_i++) {
		Light_readTo(file, &(result->lights[item_i]));
	}

	// Read the materials
	fscanf(file, "MATERIALS %u\n", &(result->n_materials));
	result->materials = (material_t*) malloc(sizeof(material_t) * result->n_materials);
	for (item_i = 0; item_i < result->n_materials; item_i++) {
		Material_readTo(file, &(result->materials[item_i]));
	}

	// Read the objects
	fscanf(file, "OBJECTS %u\n", &(result->n_objects));
	result->objects = (object_t*) malloc(sizeof(object_t) * result->n_objects);
	for (item_i = 0; item_i < result->n_objects; item_i++) {
		Object_readTo(file, &(result->objects[item_i]));
	}

	return result;
}

// Copies the specified world to the device
world_t* World_toDevice (world_t* source)
{
	world_t* final;
	world_t* result;

	// Create temporary data to correct pointers on device
	result = (world_t*) malloc(sizeof(world_t));
	memcpy(result, source, sizeof(world_t));

	// Allocare space for the world objects
	hipMalloc(&(result->lights), sizeof(light_t) * source->n_lights);
	hipMalloc(&(result->materials), sizeof(material_t) * source->n_materials);
	hipMalloc(&(result->objects), sizeof(object_t) * source->n_objects);
	// Copy the world object data to the device
	hipMemcpy(result->lights, source->lights, sizeof(light_t) * source->n_lights, hipMemcpyHostToDevice);
	hipMemcpy(result->materials, source->materials, sizeof(material_t) * source->n_materials, hipMemcpyHostToDevice);
	hipMemcpy(result->objects, source->objects, sizeof(object_t) * source->n_objects, hipMemcpyHostToDevice);

	// Allocate space for the world on the device
	hipMalloc(&final, sizeof(world_t));
	// Copy the world data to the device
	hipMemcpy(final, result, sizeof(world_t), hipMemcpyHostToDevice);
	// Free the resources allocated for the temporary result
	free(result);

	return final;
}

// Frees resources allocated for a world on the host
void World_freeHost (world_t* world)
{
	// Free memory allocated for lights
	free(world->lights);
	// Free memory allocated for materials
	free(world->materials);
	// Free memory allocated for objects
	free(world->objects);
	// Free memory allocated for world object
	free(world);
}

// Frees resources allocated for a world on the device
void World_freeDevice (world_t* world)
{
	// Copy the world object back to host so we can read array locations
	world_t* temp = (world_t*) malloc(sizeof(world_t));
	hipMemcpy(&temp, &world, sizeof(world_t), hipMemcpyDeviceToHost);
	// Free memory allocated for lights
	hipFree(temp->lights);
	// Free memory allocated for materials
	hipFree(temp->materials);
	// Free memory allocated for objects
	hipFree(temp->objects);
	// Free memory allocated for world object
	hipFree(world);
	// Free temporary memory to get device address locations
	free(temp);
}
