
#include "world.h"

// Reads the details of a world from the specified file
world_t * World_read (FILE * file)
{
	int item_i;
	// Allocate space for the result
	world_t * result = (world_t *) malloc(sizeof(world_t));

	// Read background color of the world
	fscanf(file, "BACKGROUND (%hhu, %hhu, %hhu)\n",
		&(result->bg[R]), &(result->bg[G]), &(result->bg[B]));
	// Read global ambient brightness
	fscanf(file, "AMBIENT %f\n", &(result->global_ambient));

	// Read the lights
	fscanf(file, "LIGHTS %u\n", &(result->n_lights));
	result->lights = (light_t *) malloc(sizeof(light_t) * result->n_lights);
	for (item_i = 0; item_i < result->n_lights; item_i++) {
		Light_readTo(file, &(result->lights[item_i]));
	}

	// Read the materials
	fscanf(file, "MATERIALS %u\n", &(result->n_materials));
	result->materials = (material_t *)malloc(sizeof(material_t) * result->n_materials);
	for (item_i = 0; item_i < result->n_materials; item_i++) {
		Material_readTo(file, &(result->materials[item_i]));
	}

	// Read the objects
	fscanf(file, "OBJECTS %u\n", &(result->n_objects));
	result->objects = (object_t *) malloc(sizeof(object_t) * result->n_objects);
	for (item_i = 0; item_i < result->n_objects; item_i++) {
		Object_readTo(file, &(result->objects[item_i]));
	}

	return result;
}

// Copies the specified world to the device
world_t * World_toDevice (world_t * source, int * size)
{
	world_t * final, 
			* result;

	int w_size = sizeof(world_t),
		l_size = sizeof(light_t) * source->n_lights,
		m_size = sizeof(material_t) * source->n_materials,
		o_size = sizeof(object_t) * source->n_objects;

	// Create temporary data to correct pointers on device
	result = (world_t *)malloc(w_size);
	memcpy(result, source, w_size);

	// Allocare space for the world objects
	hipMalloc(&(result->lights), l_size);
	hipMalloc(&(result->materials), m_size);
	hipMalloc(&(result->objects), o_size);
	// Copy the world object data to the device
	hipMemcpy(result->lights, source->lights, l_size, hipMemcpyHostToDevice);
	hipMemcpy(result->materials, source->materials, m_size, hipMemcpyHostToDevice);
	hipMemcpy(result->objects, source->objects, o_size, hipMemcpyHostToDevice);

	// Allocate space for the world on the device
	hipMalloc(&final, sizeof(world_t));
	// Copy the world data to the device
	hipMemcpy(final, result, sizeof(world_t), hipMemcpyHostToDevice);
	// Free the resources allocated for the temporary result
	free(result);

	*size = w_size + l_size + m_size + o_size;
	return final;
}

// Copies the specified world to the device's shared memory
__device__ world_t * World_toShared (void * smem, world_t * source)
{
	uint8_t * world_bytes = (uint8_t *) smem;
	int w_size = sizeof(world_t),
		l_size = sizeof(light_t) * source->n_lights,
		m_size = sizeof(material_t) * source->n_materials,
		o_size = sizeof(object_t) * source->n_objects;

	world_t * result = (world_t *) world_bytes;

	memcpy(world_bytes, source, w_size);

	// Set memory addresses
	result->lights = (light_t *)(world_bytes + w_size);
	result->materials = (material_t *)(world_bytes + w_size + l_size);
	result->objects = (object_t *)(world_bytes + w_size + l_size + m_size);

	// Copy the world object data to the device
	memcpy(result->lights, source->lights, l_size);
	memcpy(result->materials, source->materials, m_size);
	memcpy(result->objects, source->objects, o_size);

	return result;
}

// Frees resources allocated for a world on the host
void World_freeHost (world_t * world)
{
	// Free memory allocated for lights
	free(world->lights);
	// Free memory allocated for materials
	free(world->materials);
	// Free memory allocated for objects
	free(world->objects);
	// Free memory allocated for world object
	free(world);
}

// Frees resources allocated for a world on the device
void World_freeDevice (world_t * world)
{
	// Copy the world object back to host so we can read array locations
	world_t * temp = (world_t *) malloc(sizeof(world_t));
	hipMemcpy(temp, world, sizeof(world_t), hipMemcpyDeviceToHost);
	// Free memory allocated for lights
	hipFree(temp->lights);
	// Free memory allocated for materials
	hipFree(temp->materials);
	// Free memory allocated for objects
	hipFree(temp->objects);
	// Free memory allocated for world object
	hipFree(world);
	// Free temporary memory to get device address locations
	free(temp);
}
